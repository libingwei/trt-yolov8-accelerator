#include "hip/hip_runtime.h"
#include "decode_yolo_plugin.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>

// Simple CUDA kernel: convert [N, C] raw head [x,y,w,h,obj,cls...] or [x1,y1,x2,y2,conf,cls]
// into [N,6] with [x1,y1,x2,y2,conf,cls] without NMS. This mirrors the CPU decode in trt_utils.
// Note: This kernel assumes a single output tensor and does not apply letterbox reverse mapping.
// Mapping back to original image space should be done outside if needed (requires pad/scale per-sample).

namespace {
__device__ __forceinline__ float clampf(float v, float lo, float hi){ return fminf(fmaxf(v, lo), hi); }

// Enhanced kernel with per-sample letterbox inverse mapping
__global__ void decode_kernel_advanced(const float* __restrict__ in, int N, int C,
                                      const float* __restrict__ letterboxParams, // [N, 4]: padX, padY, scale, origScale per sample
                                      int hasObj, int numClasses, int coordsIsXYWH,
                                      float confTh, int netW, int netH,
                                      float* __restrict__ out){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=N) return;
    
    const float* p = in + i*C;
    float x = p[0], y = p[1], w = p[2], h = p[3];
    float obj = hasObj ? p[4] : 1.f;
    int clsBest = 0; float clsScore = 0.f;
    int clsStart = hasObj ? 5 : 4;
    for(int k=0;k<numClasses && (clsStart+k)<C; ++k){ 
        float v = p[clsStart+k]; 
        if(v>clsScore){ clsScore=v; clsBest=k; } 
    }
    float conf = obj * clsScore;
    if(conf < confTh){ 
        out[i*6+0]=0; out[i*6+1]=0; out[i*6+2]=0; out[i*6+3]=0; out[i*6+4]=0; out[i*6+5]=-1; 
        return; 
    }
    
    float x1, y1, x2, y2;
    if(coordsIsXYWH){
        x1 = x - w*0.5f; y1 = y - h*0.5f; x2 = x + w*0.5f; y2 = y + h*0.5f;
    } else {
        x1 = x; y1 = y; x2 = w; y2 = h;
    }
    
    // Apply letterbox inverse mapping if params provided
    if(letterboxParams != nullptr) {
        const float* params = letterboxParams + i*4; // [padX, padY, scale, origScale]
        float padX = params[0], padY = params[1], scale = params[2];
        
        // Map from network coords to original image coords
        x1 = (x1 - padX) / scale;
        y1 = (y1 - padY) / scale;
        x2 = (x2 - padX) / scale;
        y2 = (y2 - padY) / scale;
        
        // Clamp to valid range (assuming origW/origH not known, use large bounds)
        x1 = clampf(x1, 0.f, 10000.f);
        y1 = clampf(y1, 0.f, 10000.f);
        x2 = clampf(x2, 0.f, 10000.f);
        y2 = clampf(y2, 0.f, 10000.f);
    }
    
    out[i*6+0] = x1; out[i*6+1] = y1; out[i*6+2] = x2; out[i*6+3] = y2; 
    out[i*6+4] = conf; out[i*6+5] = (float)clsBest;
}

// Simple kernel for backward compatibility (no letterbox)
__global__ void decode_kernel(const float* __restrict__ in, int N, int C,
                              int hasObj, int numClasses, int coordsIsXYWH,
                              float confTh,
                              float* __restrict__ out){
    decode_kernel_advanced(in, N, C, nullptr, hasObj, numClasses, coordsIsXYWH, confTh, 640, 640, out);
}
}

using namespace nvinfer1;

nvinfer1::DimsExprs DecodeYoloPlugin::getOutputDimensions(int, const nvinfer1::DimsExprs* inputs, int nbInputs, nvinfer1::IExprBuilder& eb) noexcept {
    DimsExprs out;
    out.nbDims = 2;
    if(nbInputs < 1){ out.d[0] = eb.constant(0); out.d[1] = eb.constant(6); return out; }
    
    const auto& in = inputs[0]; // YOLO head input
    if(in.nbDims==2){ out.d[0] = in.d[0]; }
    else if(in.nbDims==3){ out.d[0] = in.d[1]; }
    else { out.d[0] = eb.constant(0); }
    out.d[1] = eb.constant(6);
    return out;
}

int DecodeYoloPlugin::enqueue(const nvinfer1::PluginTensorDesc* inputDescs,
                              const nvinfer1::PluginTensorDesc* outputDescs,
                              const void* const* inputs,
                              void* const* outputs,
                              void*, hipStream_t stream) noexcept {
    const auto& inDesc = inputDescs[0];
    int nbDims = inDesc.dims.nbDims;
    int N = 0, C = 0;
    if(nbDims==2){ N = inDesc.dims.d[0]; C = inDesc.dims.d[1]; }
    else if(nbDims==3){ N = inDesc.dims.d[1]; C = inDesc.dims.d[2]; }
    else { return 1; }

    const float* in = static_cast<const float*>(inputs[0]);
    const float* letterboxParams = nullptr;
    
    // Check if letterbox parameters are provided
    if(mUseLetterbox && inputDescs->dims.nbDims >= 2) {
        letterboxParams = static_cast<const float*>(inputs[1]);
    }
    
    float* out = static_cast<float*>(outputs[0]);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    
    if(letterboxParams) {
        decode_kernel_advanced<<<blocks, threads, 0, stream>>>(
            in, N, C, letterboxParams, /*hasObj*/1, /*numClasses*/(C-5), /*coordsIsXYWH*/1, 
            /*confTh*/0.0f, /*netW*/640, /*netH*/640, out);
    } else {
        decode_kernel<<<blocks, threads, 0, stream>>>(
            in, N, C, /*hasObj*/1, /*numClasses*/(C-5), /*coordsIsXYWH*/1, /*confTh*/0.0f, out);
    }
    return 0;
}

extern "C" void registerDecodeYoloPlugin(){
    static DecodeYoloPluginCreator creator;
    auto* reg = getPluginRegistry();
    if(reg) { 
        // Register with empty namespace (default)
        reg->registerCreator(creator, "");
        // Optionally set a custom namespace if needed
        creator.setPluginNamespace(""); 
    }
}
